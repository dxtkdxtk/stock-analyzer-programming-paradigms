#include "SACudaLibrary.h"
#include "SACudaKernel.h"

extern "C" int TestCudaAdd(int a, int b)
{
	int * d_result;
	int result;

	hipMalloc((void **)&d_result, sizeof(int));

	TestCudaAddKernel<<<1,1>>>(a, b, d_result);

	hipMemcpy((void *)&result, (void *)d_result, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_result);

	return result;
}

